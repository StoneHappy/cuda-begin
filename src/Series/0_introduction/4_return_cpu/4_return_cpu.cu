#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
__global__ void kernel(int *pret) {
    *pret = 42;
}

int main( int argc, char** argv )
{
    // hipMalloc
    int *pret;
    checkCudaErrors(hipMalloc(&pret, sizeof(int)));
    kernel<<<1,1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    int ret;
    checkCudaErrors(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
    printf("ret: %d\n", ret);
    hipFree(pret);

    // hipMallocManaged
    checkCudaErrors(hipMallocManaged(&pret, sizeof(int)));
    kernel<<<1,1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    printf("pret: %d\n", *pret);
    hipFree(pret);
    return 0;
}