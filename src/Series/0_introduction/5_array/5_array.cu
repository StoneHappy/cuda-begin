#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel0(int *arr, int n) {
    int i = threadIdx.x;
    arr[i] = i;
}

__global__ void kernel1(int *arr, int n) {
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        arr[i] = i;
    }
}

__global__ void kernel2(int *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        arr[i] = i;
    }
}

int main( int argc, char** argv )
{
    int n = 32;
    int *arr;

    // set fixed thread nums
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));
    kernel0<<<1,n>>>(arr, n);
    checkCudaErrors(hipDeviceSynchronize());
    for (size_t i = 0; i< n; i++)
    {
        printf("arr0[%zd]: %d\n", i, arr[i]);
    }
    hipFree(arr);

    // use blockDim auto get threadId
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));
    kernel1<<<1,4>>>(arr, n);
    checkCudaErrors(hipDeviceSynchronize());
    for (size_t i = 0; i< n; i++)
    {
        printf("arr1[%zd]: %d\n", i, arr[i]);
    }
    hipFree(arr);

    // use blockDim and gridDim auto get threadId
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));
    kernel2<<<2,16>>>(arr, n);
    checkCudaErrors(hipDeviceSynchronize());
    for (size_t i = 0; i< n; i++)
    {
        printf("arr2[%zd]: %d\n", i, arr[i]);
    }
    hipFree(arr);

    return 0;
}