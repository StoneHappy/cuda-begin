#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void kernel(){
    printf("Hi Cuda World");
}

int main( int argc, char** argv )
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}