#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__device__ void show_cuda_arch()
{
#ifdef __CUDA_ARCH__
    printf("cuda_arch: %d\n", __CUDA_ARCH__);
#endif
}

__global__ void test() {
    show_cuda_arch();
}

int main( int argc, char** argv )
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}