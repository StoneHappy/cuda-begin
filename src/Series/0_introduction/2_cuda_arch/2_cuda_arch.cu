#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__device__ void show_cuda_arch()
{
#ifdef __CUDA_ARCH__
    printf("cuda_arch: %d\n", __CUDA_ARCH__);
#endif
}

__global__ void kernel() {
    show_cuda_arch();
}

int main( int argc, char** argv )
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}