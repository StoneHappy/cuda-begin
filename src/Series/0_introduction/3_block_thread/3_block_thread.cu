#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void kernel1() {
    printf("Thread %d of %d\n", threadIdx.x, blockDim.x);
}

__global__ void kernel2() {
    printf("Block %d of %d, Thread %d of %d\n", blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);
}

__global__ void kernel3() {
    unsigned  int tid  = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned  int tnum = blockDim.x * gridDim.x;
    printf("Flattened Thread %d of %d\n", tid, tnum);
}

int main( int argc, char** argv )
{
    kernel1<<<1,3>>>();
    hipDeviceSynchronize();
    kernel2<<<2,3>>>();
    hipDeviceSynchronize();
    kernel3<<<2,3>>>();
    hipDeviceSynchronize();
    return 0;
}