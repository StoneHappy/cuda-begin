#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__device__ void say_hello()
{
    printf("hello!\n");
}

__global__ void kernel() {
    say_hello();
}

int main( int argc, char** argv )
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}