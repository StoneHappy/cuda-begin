#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__device__ void say_hello()
{
    printf("hello!\n");
}

__global__ void test() {
    say_hello();
}

int main( int argc, char** argv )
{
    test<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}